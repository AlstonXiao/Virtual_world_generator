#include "hip/hip_runtime.h"
// Modified from Ken Perlin's implementation of the Perlin Noise
// https://mrl.nyu.edu/~perlin/noise/
#include <cmath>
#include "PerlinNoise.cuh"
#include <iostream>

#ifndef GLOBAL_H
#define GLOBAL_H
#include "globals.h"
#endif

using namespace std;
const int Permutation[512] = { 151,160,137,91,90,15,
    131,13,201,95,96,53,194,233,7,225,140,36,103,30,69,142,8,99,37,240,21,10,23,
    190, 6,148,247,120,234,75,0,26,197,62,94,252,219,203,117,35,11,32,57,177,33,
    88,237,149,56,87,174,20,125,136,171,168, 68,175,74,165,71,134,139,48,27,166,
    77,146,158,231,83,111,229,122,60,211,133,230,220,105,92,41,55,46,245,40,244,
    102,143,54, 65,25,63,161, 1,216,80,73,209,76,132,187,208, 89,18,169,200,196,
    135,130,116,188,159,86,164,100,109,198,173,186, 3,64,52,217,226,250,124,123,
    5,202,38,147,118,126,255,82,85,212,207,206,59,227,47,16,58,17,182,189,28,42,
    223,183,170,213,119,248,152, 2,44,154,163, 70,221,153,101,155,167, 43,172,9,
    129,22,39,253, 19,98,108,110,79,113,224,232,178,185, 112,104,218,246,97,228,
    251,34,242,193,238,210,144,12,191,179,162,241, 81,51,145,235,249,14,239,107,
    49,192,214, 31,181,199,106,157,184, 84,204,176,115,121,50,45,127, 4,150,254,
    138,236,205,93,222,114,67,29,24,72,243,141,128,195,78,66,215,61,156,180,
    151,160,137,91,90,15,
    131,13,201,95,96,53,194,233,7,225,140,36,103,30,69,142,8,99,37,240,21,10,23,
    190, 6,148,247,120,234,75,0,26,197,62,94,252,219,203,117,35,11,32,57,177,33,
    88,237,149,56,87,174,20,125,136,171,168, 68,175,74,165,71,134,139,48,27,166,
    77,146,158,231,83,111,229,122,60,211,133,230,220,105,92,41,55,46,245,40,244,
    102,143,54, 65,25,63,161, 1,216,80,73,209,76,132,187,208, 89,18,169,200,196,
    135,130,116,188,159,86,164,100,109,198,173,186, 3,64,52,217,226,250,124,123,
    5,202,38,147,118,126,255,82,85,212,207,206,59,227,47,16,58,17,182,189,28,42,
    223,183,170,213,119,248,152, 2,44,154,163, 70,221,153,101,155,167, 43,172,9,
    129,22,39,253, 19,98,108,110,79,113,224,232,178,185, 112,104,218,246,97,228,
    251,34,242,193,238,210,144,12,191,179,162,241, 81,51,145,235,249,14,239,107,
    49,192,214, 31,181,199,106,157,184, 84,204,176,115,121,50,45,127, 4,150,254,
    138,236,205,93,222,114,67,29,24,72,243,141,128,195,78,66,215,61,156,180
};

__device__ float fade(float t) { return t * t * t * (t * (t * 6 - 15) + 10); }

__device__ float lerp(float t, float a, float b) { return a + t * (b - a); }

__device__ float grad(int hash, float x, float y, float z) {
    int h = hash & 15;                      // CONVERT LO 4 BITS OF HASH CODE
    float u = h<8 ? x : y,                 // INTO 12 GRADIENT DIRECTIONS.
           v = h<4 ? y : h==12||h==14 ? x : z;
    return ((h&1) == 0 ? u : -u) + ((h&2) == 0 ? v : -v);
}

__device__ float noise(float x, float y, float z, int* permutation) {
    int X = (int)floor(x) & 255;                  // FIND UNIT CUBE THAT
    int Y = (int)floor(y) & 255;                  // CONTAINS POINT.
    int Z = (int)floor(z) & 255;
    x -= floor(x);                                // FIND RELATIVE X,Y,Z
    y -= floor(y);                                // OF POINT IN CUBE.
    z -= floor(z);
    float u = fade(x);                                // COMPUTE FADE CURVES
    float v = fade(y);                                // FOR EACH OF X,Y,Z.
    float w = fade(z);

    int A = permutation[X  ]+Y, AA = permutation[A]+Z, AB = permutation[A+1]+Z,      // HASH COORDINATES OF
        B = permutation[X+1]+Y, BA = permutation[B]+Z, BB = permutation[B+1]+Z;      // THE 8 CUBE CORNERS,

    return lerp(w, lerp(v, lerp(u, grad(permutation[AA  ], x  , y  , z   ),  // AND ADD
                                   grad(permutation[BA  ], x-1, y  , z   )), // BLENDED
                           lerp(u, grad(permutation[AB  ], x  , y-1, z   ),  // RESULTS
                                   grad(permutation[BB  ], x-1, y-1, z   ))),// FROM  8
                   lerp(v, lerp(u, grad(permutation[AA+1], x  , y  , z-1 ),  // CORNERS
                                   grad(permutation[BA+1], x-1, y  , z-1 )), // OF CUBE
                           lerp(u, grad(permutation[AB+1], x  , y-1, z-1 ),
                                   grad(permutation[BB+1], x-1, y-1, z-1 ))));
 }



__global__ void PerlinKernel(Vector3<float>* points, const int* permu, const size_t size, const int octaves, const float persistence, const float divided) {
    __shared__ int permutation[512];
    unsigned int pos = blockIdx.x * blockDim.x + threadIdx.x;
    if (threadIdx.x < 512) permutation[threadIdx.x] = permu[threadIdx.x];
    __syncthreads();
    float addedtotal = 0;
    float frequency = 1;
    float amplitude = 1;
    float maxValue = 0;
    float x = points[pos].x;
    float y = points[pos].y;
    float z = points[pos].z;
    if (pos < size){
        for(int i=0;i<octaves;i++) {
            addedtotal += noise(x * frequency, y * frequency, z * frequency, permutation) * amplitude;
            maxValue += amplitude;
            amplitude *= persistence;
            frequency *= 2;
        }
        points[pos].y += addedtotal/maxValue / divided;
    }
}
__host__ void randomize(OutputObject Out) {
    size_t objectCount = Out.otherObjectCount + Out.streetCount;
    // cout<<"now in CUDA, with "<<objectCount<<" objects\n" << std::flush;
    Vector3<float> **host_vertex = new Vector3<float>*[objectCount];
    Vector3<float> **device_vertex = new Vector3<float>*[objectCount];
    hipStream_t stream[objectCount]; 
    // cout<<"allocate done\n" << std::flush;
    int *permu;
    // hipError_t code;
    hipMallocManaged(&permu, 512*sizeof(int));
    // cout<<hipGetErrorString(code);
    // cout<<Permutation[10]<<"\n"<< std::flush;
    // cout<<"stg"<< std::flush;
    for (int i = 0; i < 512; i++) {
        permu[i] = Permutation[i];
        // cout<<permu[i]<<"\n"<< std::flush;
    }
    // cout<<permu[2]<<" "<<permu[511]<<"\n"<< std::flush;
    // cout<<Out.objects[0].vertice_count<<"\n"<< std::flush;
    for(size_t i = 0; i < objectCount; ++i) {
        hipStreamCreate(&stream[i]);
        hipHostAlloc(&host_vertex[i], Out.objects[i].vertice_count * sizeof(Vector3<float>), hipHostMallocDefault);
        hipMalloc(&device_vertex[i],  Out.objects[i].vertice_count * sizeof(Vector3<float>));
        for (size_t j = 0; j < Out.objects[i].vertice_count; j++){
            host_vertex[i][j] = Out.objects[i].vertices[j];
        }
    }
    // cout<<host_vertex[0][2]<<" "<<host_vertex[0][9999]<<"\n"<< std::flush;;

    for (size_t i = 0; i < objectCount; ++i) {
        hipMemcpyAsync( device_vertex[i], host_vertex[i], Out.objects[i].vertice_count * sizeof(Vector3<float>), hipMemcpyHostToDevice, stream[i]);
        unsigned int blockNum = (Out.objects[i].vertice_count + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK;
        int octaves = i < Out.streetCount? 4 : 2;
        float persistence = i < Out.streetCount? 0.75 : 0.5;
        float divided = i < Out.streetCount? 20 : 40;
        PerlinKernel<<<blockNum, THREAD_PER_BLOCK, 0, stream[i]>>>(device_vertex[i],permu, Out.objects[i].vertice_count, octaves, persistence, divided);
        hipMemcpyAsync( host_vertex[i], device_vertex[i], Out.objects[i].vertice_count * sizeof(Vector3<float>), hipMemcpyDeviceToHost, stream[i]);
    }
    for(size_t i = 0; i < objectCount; ++i) hipStreamSynchronize( stream[i]);
    // cout<<host_vertex[0][2]<<" "<<host_vertex[0][9999]<<"\n"<< std::flush;;

    for(size_t i = 0; i < objectCount; ++i) {
        for (size_t j = 0; j < Out.objects[i].vertice_count; j++){
            Out.objects[i].vertices[j] = host_vertex[i][j];
        }  
        hipStreamDestroy(stream[i]);
        hipHostFree(host_vertex[i]);
        hipFree(device_vertex[i]);
    }
}